#include <hip/hip_runtime.h>
#include <iostream>

__global__ void helloFromGPU() {
    printf("Hello from GPU!\n");
}

int main() {
    // Get CUDA device properties
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    std::cout << "Number of CUDA devices: " << deviceCount << "\n";

    for (int i = 0; i < deviceCount; ++i) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);

        std::cout << "Device " << i << ": " << prop.name << "\n";
        std::cout << "  Compute capability: " << prop.major << "." << prop.minor << "\n";
        std::cout << "  Total global memory: " << prop.totalGlobalMem / (1024 * 1024) << " MB\n";
    }

    // Launch a simple kernel
    helloFromGPU<<<1, 1>>>();
    hipDeviceSynchronize();

    return 0;
}